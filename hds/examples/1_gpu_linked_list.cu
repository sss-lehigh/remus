#include "hip/hip_runtime.h"
#include <cstdio>

#include <rome/hds/allocator/allocator.h>
#include <rome/hds/linked_list/lock_linked_list.h>
#include <rome/hds/linked_list/locked_nodes/reg_cached_nodes.h>
#include <rome/hds/threadgroup/threadgroup.h>
#include <set>

HDS_HOST_DEVICE void error() {
#if defined(__CUDA_ARCH__)
__trap();
#else
exit(1);
#endif
}

#define ASSERT(x, y) if(!(x)) { printf("%s did not evaluate to true for i = %d\n", #x, (y)); error(); }

__global__ void single_thread_test(rome::hds::lock_linked_list<int, 2, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>* ll) {

  ll = new (ll) rome::hds::lock_linked_list<int, 2, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>(); 

  auto group = rome::hds::threadgroup::single_threadgroup{};

  ASSERT(!ll->contains(1, group), 1);

  for (int i = 0; i < 100; ++i) {
    int r = i;

    ASSERT(ll->insert(r, group), r);

    if (!ll->validate(group)) {
      error();
    }
  }
  
  for (int i = 0; i < 100; ++i) {
    int r = i;

    ASSERT(ll->remove(r, group), r);

    if(!ll->validate(group)) {
      error();
    }
  }

  ll->~lock_linked_list<int, 2, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>(); 
}

__global__ void warp_test(rome::hds::lock_linked_list<int, 32, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>* ll) {

  auto warp = rome::hds::threadgroup::warp_threadgroup{};
  if (warp.is_leader()) {
    new (ll) rome::hds::lock_linked_list<int, 32, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>(); 
  }
  warp.sync();

  static_assert(decltype(warp)::size == 32);

  ASSERT(!ll->contains(1, warp), 1);

  for(int i = 0; i < 100; ++i) {
    int r = i;

    ASSERT(ll->insert(r, warp), r);

    if(!ll->validate(warp)) {
      error();
    }
  }

  for(int i = 0; i < 100; ++i) {
    int r = i;

    ASSERT(ll->remove(r, warp), r);

    if(!ll->validate(warp)) {
      error();
    }
  }

  warp.sync();
  if (warp.is_leader()) {
    ll->~lock_linked_list<int, 32, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>(); 
  }
}

int main() {
  rome::hds::lock_linked_list<int, 2, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator> ll;
  auto group = rome::hds::threadgroup::single_threadgroup{};
  ASSERT(!ll.contains(1, group), 1);

  std::set<int> reference;

  for(int i = 0; i < 100; ++i) {

    if(rand() % 2 == 0) {

      int r = rand();

      bool inserted = reference.insert(r).second;
      ASSERT(ll.insert(r, group) == inserted, r);

      printf("\nInserted %d\n", r);
      ll.print(group);

    } else {

      int r = rand();

      bool removed = (reference.erase(r) == 1);

      ASSERT(ll.remove(r, group) == removed, r);

      printf("\nRemoved %d\n", r);
      ll.print(group);

    }

    if(!ll.validate(group)) {
      return 1;
    }

    for(auto elm : reference) {
      ASSERT(ll.contains(elm, group), elm);
    }
  }

  rome::hds::allocator::device_allocator dev_mem;
  rome::hds::lock_linked_list<int, 2, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>* st_gpu_ll;
  st_gpu_ll = dev_mem.allocate<rome::hds::lock_linked_list<int, 2, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>>(1);

  single_thread_test<<<1, 1>>>(st_gpu_ll);

  auto err = hipDeviceSynchronize();

  if(err != hipSuccess) {
    throw std::runtime_error(std::string(hipGetErrorName(err)) + " : " + std::string(hipGetErrorString(err)));
  }

  dev_mem.deallocate(st_gpu_ll, 1);

  rome::hds::lock_linked_list<int, 32, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>* w_gpu_ll;
  w_gpu_ll = dev_mem.allocate<rome::hds::lock_linked_list<int, 32, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>>(1);

  warp_test<<<1, 32>>>(w_gpu_ll);

  err = hipDeviceSynchronize();

  if(err != hipSuccess) {
    throw std::runtime_error(std::string(hipGetErrorName(err)) + " : " + std::string(hipGetErrorString(err)));
  }
  
  dev_mem.deallocate(w_gpu_ll, 1);

  return 0;
}

__launch_bounds__(1024, 1)
__global__ void warp_insert(rome::hds::lock_linked_list<int, 32, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>* ll, int i) {

  auto warp = rome::hds::threadgroup::warp_threadgroup{};

  static_assert(decltype(warp)::size == 32);

  ll->insert(i, warp);

}

__launch_bounds__(1024, 1)
__global__ void warp_remove(rome::hds::lock_linked_list<int, 32, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>* ll, int i) {

  auto warp = rome::hds::threadgroup::warp_threadgroup{};

  static_assert(decltype(warp)::size == 32);

  ll->remove(i, warp);

}

__global__ void warp_contains(rome::hds::lock_linked_list<int, 32, rome::hds::locked_nodes::reg_cached_node_pointer, rome::hds::allocator::heap_allocator>* ll, int i) {

  auto warp = rome::hds::threadgroup::warp_threadgroup{};

  static_assert(decltype(warp)::size == 32);

  ll->contains(i, warp);
 
}
