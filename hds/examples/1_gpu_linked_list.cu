#include "hip/hip_runtime.h"
#include <cstdio>

#include <remus/hds/allocator/allocator.h>
#include <remus/hds/linked_list/lock_linked_list.h>
#include <remus/hds/linked_list/locked_nodes/reg_cached_nodes.h>
#include <remus/hds/threadgroup/threadgroup.h>
#include <set>

HDS_HOST_DEVICE void error() {
#if defined(__CUDA_ARCH__)
__trap();
#else
exit(1);
#endif
}

#define ASSERT(x, y) if(!(x)) { printf("%s did not evaluate to true for i = %d\n", #x, (y)); error(); }

__global__ void single_thread_test(remus::hds::lock_linked_list<int, 2, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>* ll) {

  ll = new (ll) remus::hds::lock_linked_list<int, 2, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>(); 

  auto group = remus::hds::threadgroup::single_threadgroup{};

  ASSERT(!ll->contains(1, group), 1);

  for (int i = 0; i < 100; ++i) {
    int r = i;

    ASSERT(ll->insert(r, group), r);

    if (!ll->validate(group)) {
      error();
    }
  }
  
  for (int i = 0; i < 100; ++i) {
    int r = i;

    ASSERT(ll->remove(r, group), r);

    if(!ll->validate(group)) {
      error();
    }
  }

  ll->~lock_linked_list<int, 2, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>(); 
}

__global__ void warp_test(remus::hds::lock_linked_list<int, 32, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>* ll) {

  auto warp = remus::hds::threadgroup::warp_threadgroup{};
  if (warp.is_leader()) {
    new (ll) remus::hds::lock_linked_list<int, 32, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>(); 
  }
  warp.sync();

  static_assert(decltype(warp)::size == 32);

  ASSERT(!ll->contains(1, warp), 1);

  for(int i = 0; i < 100; ++i) {
    int r = i;

    ASSERT(ll->insert(r, warp), r);

    if(!ll->validate(warp)) {
      error();
    }
  }

  for(int i = 0; i < 100; ++i) {
    int r = i;

    ASSERT(ll->remove(r, warp), r);

    if(!ll->validate(warp)) {
      error();
    }
  }

  warp.sync();
  if (warp.is_leader()) {
    ll->~lock_linked_list<int, 32, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>(); 
  }
}

int main() {
  remus::hds::lock_linked_list<int, 2, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator> ll;
  auto group = remus::hds::threadgroup::single_threadgroup{};
  ASSERT(!ll.contains(1, group), 1);

  std::set<int> reference;

  for(int i = 0; i < 100; ++i) {

    if(rand() % 2 == 0) {

      int r = rand();

      bool inserted = reference.insert(r).second;
      ASSERT(ll.insert(r, group) == inserted, r);

      printf("\nInserted %d\n", r);
      ll.print(group);

    } else {

      int r = rand();

      bool removed = (reference.erase(r) == 1);

      ASSERT(ll.remove(r, group) == removed, r);

      printf("\nRemoved %d\n", r);
      ll.print(group);

    }

    if(!ll.validate(group)) {
      return 1;
    }

    for(auto elm : reference) {
      ASSERT(ll.contains(elm, group), elm);
    }
  }

  remus::hds::allocator::device_allocator dev_mem;
  remus::hds::lock_linked_list<int, 2, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>* st_gpu_ll;
  st_gpu_ll = dev_mem.allocate<remus::hds::lock_linked_list<int, 2, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>>(1);

  single_thread_test<<<1, 1>>>(st_gpu_ll);

  auto err = hipDeviceSynchronize();

  if(err != hipSuccess) {
    throw std::runtime_error(std::string(hipGetErrorName(err)) + " : " + std::string(hipGetErrorString(err)));
  }

  dev_mem.deallocate(st_gpu_ll, 1);

  remus::hds::lock_linked_list<int, 32, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>* w_gpu_ll;
  w_gpu_ll = dev_mem.allocate<remus::hds::lock_linked_list<int, 32, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>>(1);

  warp_test<<<1, 32>>>(w_gpu_ll);

  err = hipDeviceSynchronize();

  if(err != hipSuccess) {
    throw std::runtime_error(std::string(hipGetErrorName(err)) + " : " + std::string(hipGetErrorString(err)));
  }
  
  dev_mem.deallocate(w_gpu_ll, 1);

  return 0;
}

__launch_bounds__(1024, 1)
__global__ void warp_insert(remus::hds::lock_linked_list<int, 32, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>* ll, int i) {

  auto warp = remus::hds::threadgroup::warp_threadgroup{};

  static_assert(decltype(warp)::size == 32);

  ll->insert(i, warp);

}

__launch_bounds__(1024, 1)
__global__ void warp_remove(remus::hds::lock_linked_list<int, 32, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>* ll, int i) {

  auto warp = remus::hds::threadgroup::warp_threadgroup{};

  static_assert(decltype(warp)::size == 32);

  ll->remove(i, warp);

}

__global__ void warp_contains(remus::hds::lock_linked_list<int, 32, remus::hds::locked_nodes::reg_cached_node_pointer, remus::hds::allocator::heap_allocator>* ll, int i) {

  auto warp = remus::hds::threadgroup::warp_threadgroup{};

  static_assert(decltype(warp)::size == 32);

  ll->contains(i, warp);
 
}
